
#include <hip/hip_runtime.h>

__device__
void ghhl_add(const float* jac, const float* hes, float res, float lambda, float* h, float* hl, float* g, int tid, int Nprobs)
{
	int l = 0;
	for (int j = 0; j < 4; ++j) {
		// this sums up hessian parts
		for (int k = 0; k <= j; ++k) {
			int lidx = l*Nprobs+tid;
			float jtemp = jac[j] * jac[k];
			float hjtemp = hes[l] + jtemp;
			h[lidx] += hjtemp;
			if (j != k) {
				hl[lidx] += hjtemp;
			} else {
				hl[lidx] += hjtemp + fmaxf(lambda*jtemp, 1e-4f);
			}
			++l;
		}

		g[j*Nprobs+tid] += jac[j] * res;
	}
}


__device__
void enc_to_vel_fghhl(const float* params, const float* consts, const float* data, const float* lam,
	float* f, float* g, float* h, float* hl, int tid, int Nprobs)
{
	float res;
	float jac[4];
	float hes[10];
	float lambda = lam[tid];

	float M0 = params[tid];
	float vx = params[Nprobs+tid];
	float vy = params[2*Nprobs+tid];
	float vz = params[3*Nprobs+tid];

	float k = 1.813799f;

	float vt[8];
	// Encoding 1
	vt[0] = cosf(-k*(vx+vy+vz));
	vt[1] = sinf(-k*(vx+vy+vz));
	// Encoding 2
	vt[2] = cosf(k*(vx+vy-vz));
	vt[3] = sinf(k*(vx+vy-vz));
	// Encoding 3
	vt[4] = cosf(k*(vx-vy+vz));
	vt[5] = sinf(k*(vx-vy+vz));
	// Encoding 4
	vt[6] = cosf(k*(vy+vz-vx));
	vt[7] = sinf(k*(vy+vz-vx));

	
	// Encoding 0
	res = M0 - data[tid];
	f[tid] += res * res;

	jac[0] = 1.0f;
	jac[1] = 0.0f;
	jac[2] = 0.0f;
	jac[3] = 0.0f;

	for (int i = 0; i < 10; ++i) {
		hes[i] = 0.0f;
	}

	// Encoding 1 Real
	res = M0*vt[0] - data[Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[0];
	jac[1] = M0*vt[1]*k;
	jac[2] = jac[1];
	jac[3] = jac[1];

	hes[0] = 0.0f;
	hes[1] = vt[1]*k*res;
	hes[2] = -M0*vt[0]*k*k*res;
	hes[3] = hes[1];
	hes[4] = hes[2];
	hes[5] = hes[2];
	hes[6] = hes[1];
	hes[7] = hes[2];
	hes[8] = hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);


	// Encoding 1 Imag
	res = M0*vt[1] - data[2*Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[1];
	jac[1] = -M0*vt[0]*k;
	jac[2] = jac[1];
	jac[3] = jac[1];

	hes[0] = 0.0f;
	hes[1] = -vt[0]*k*res;
	hes[2] = -M0*vt[1]*k*k*res;
	hes[3] = hes[1];
	hes[4] = hes[2];
	hes[5] = hes[2];
	hes[6] = hes[1];
	hes[7] = hes[2];
	hes[8] = hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);

	
	// Encoding 2 Real
	res = M0*vt[2] - data[3*Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[2];
	jac[1] = -M0*vt[3]*k;
	jac[2] = jac[1];
	jac[3] = -jac[1];

	hes[0] = 0.0f;
	hes[1] = -vt[3]*k*res;
	hes[2] = -M0*vt[2]*k*k*res;
	hes[3] = hes[1];
	hes[4] = hes[2];
	hes[5] = hes[2];
	hes[6] = -hes[1];
	hes[7] = -hes[2];
	hes[8] = -hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);


	// Encoding 2 Imag
	res = M0*vt[3] - data[4*Nprobs+tid];
	f[tid] += res * res;
	
	jac[0] = vt[3];
	jac[1] = M0*vt[2]*k;
	jac[2] = jac[1];
	jac[3] = -jac[1];

	hes[0] = 0.0f;
	hes[1] = vt[2]*k*res;
	hes[2] = -M0*vt[3]*k*k*res;
	hes[3] = hes[1];
	hes[4] = hes[2];
	hes[5] = hes[2];
	hes[6] = -hes[1];
	hes[7] = -hes[2];
	hes[8] = -hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);

	// Encoding 3 Real
	res = M0*vt[4] - data[5*Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[4];
	jac[1] = M0*vt[5]*k;
	jac[2] = -jac[1];
	jac[3] = jac[1];

	hes[0] = 0.0f;
	hes[1] = vt[5]*k*res;
	hes[2] = -M0*vt[4]*k*k*res;
	hes[3] = -hes[1];
	hes[4] = -hes[2];
	hes[5] = hes[2];
	hes[6] = hes[1];
	hes[7] = hes[2];
	hes[8] = -hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);


	// Encoding 3 Imag
	res = M0*vt[5] - data[6*Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[5];
	jac[1] = M0*vt[6]*k;
	jac[2] = -jac[1];
	jac[3] = jac[1];

	hes[0] = 0.0f;
	hes[1] = vt[6]*k*res;
	hes[2] = -M0*vt[5]*k*k*res;
	hes[3] = -hes[1];
	hes[4] = -hes[2];
	hes[5] = hes[2];
	hes[6] = hes[1];
	hes[7] = hes[2];
	hes[8] = -hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);


	// Encoding 4 Real
	res = M0*vt[6] - data[7*Nprobs+tid];
	f[tid] += res * res;

	jac[0] = vt[6];
	jac[1] = M0*vt[7]*k;
	jac[2] = -jac[1];
	jac[3] = -jac[1];

	hes[0] = 0.0f;
	hes[1] = vt[7]*k*res;
	hes[2] = -M0*vt[6]*k*k*res;
	hes[3] = -hes[1];
	hes[4] = -hes[2];
	hes[5] = hes[2];
	hes[6] = -hes[1];
	hes[7] = -hes[2];
	hes[8] = hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);

	
	// Encoding 4 Imag
	res = M0*vt[7] - data[8*Nprobs+tid];
	f[tid] += res * res;
	
	jac[0] = vt[7];
	jac[1] = -M0*vt[6]*k;
	jac[2] = -jac[1];
	jac[3] = -jac[1];

	hes[0] = 0.0f;
	hes[1] = -vt[6]*k*res;
	hes[2] = -M0*vt[7]*k*k*res;
	hes[3] = -hes[1];
	hes[4] = -hes[2];
	hes[5] = hes[2];
	hes[6] = -hes[1];
	hes[7] = -hes[2];
	hes[8] = hes[2];
	hes[9] = hes[2];

	ghhl_add(jac, hes, res, lambda, h, hl, g, tid, Nprobs);
	
}

__device__ 
void enc_to_vel_f(const float* params, const float* consts, const float* data, 
	float* f, int tid, int Nprobs)
{
	float M0 = params[tid];
	float vx = params[Nprobs+tid];
	float vy = params[2*Nprobs+tid];
	float vz = params[3*Nprobs+tid];

	float k = 1.813799f;

	float vel_term;
	float res;

	// Encoding 0
	res = M0 - data[tid];
	f[tid] += res * res;

	// Encoding 1
	vel_term = -k*(vx+vy+vz);
	// Real
	res = M0*cosf(vel_term) - data[Nprobs+tid];
	f[tid] += res * res;
	// Imag
	res = M0*sinf(vel_term) - data[2*Nprobs+tid];
	f[tid] += res * res;

	// Encoding 2
	vel_term = k*(vx+vy-vz);
	// Real
	res = M0*cosf(vel_term) - data[3*Nprobs+tid];
	f[tid] += res * res;
	// Imag
	res = M0*sinf(vel_term) - data[4*Nprobs+tid];
	f[tid] += res * res;
	
	// Encoding 3
	vel_term = k*(vx-vy+vz);
	// Real
	res = M0*cosf(vel_term) - data[5*Nprobs+tid];
	f[tid] += res * res;
	// Imag
	res = M0*sinf(vel_term) - data[6*Nprobs+tid];
	f[tid] += res * res;

	// Encoding 4
	vel_term = k*(vy+vz-vx);
	// Real
	res = M0*cosf(vel_term) - data[7*Nprobs+tid];
	f[tid] += res * res;
	// Imag
	res = M0*sinf(vel_term) - data[8*Nprobs+tid];
	f[tid] += res * res;
	
}

__device__
int max_diag_abs_4_f(const float* mat, int offset) 
{
	float max_abs = -1.0f;
	int max_index = 0;
	for (int i = offset; i < 4; ++i) {
		if (fabsf(mat[i*4+i]) > max_abs) {
			max_index = i;
		}
	}
	return max_index;
}

__device__
void row_interchange_i_4_f(float* mat, int ii, int jj) 
{
	for (int k = 0; k < 4; ++k) {
		int ikn = ii*4+k;
		int jkn = jj*4+k;

		float temp;
		temp = mat[ikn];
		mat[ikn] = mat[jkn];
		mat[jkn] = temp;
	}
}

__device__
void col_interchange_i_4_f(float* mat, int ii, int jj) 
{
	for (int k = 0; k < 4; ++k) {
		int kin = k*4+ii;
		int kjn = k*4+jj;

		float temp;
		temp = mat[kin];
		mat[kin] = mat[kjn];
		mat[kjn] = temp;
	}
}

__device__
void diag_pivot_4_f(float* mat, int* perm) 
{
	for (int i = 0; i < 4; ++i) {
		perm[i] = i;
	}
	for (int i = 0; i < 4; ++i) {
		int max_abs = max_diag_abs_4_f(mat, i);
		row_interchange_i_4_f(mat, i, max_abs);
		col_interchange_i_4_f(mat, i, max_abs);
		int temp = perm[i];
		perm[i] = perm[max_abs];
		perm[max_abs] = temp;
	}
}

__device__
void gmw81_4_f(float* mat) 
{
	float t0;
	float t1 = 0.0f; // gamma
	float t2 = 0.0f; // nu
	float beta2 = 2e-7;
	float delta = 2e-7;

	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j <= i; ++j) {
			t0 = fabsf(mat[i*4+j]);
			if (i == j) {
				if (t0 > t1)
					t1 = t0;
			} else {
				if (t0 > t2)
					t2 = t0;
			}
		}
	}

	if (4 > 1) {
		t2 /= sqrtf(4*4 - 1);
	}

	if (beta2 < t1)
		beta2 = t1;
	if (beta2 < t2)
		beta2 = t2;
	t0 = t1 + t2;
	if (t0 > 1.0f)
		delta *= t0;
	// delta = eps*max(gamma + nu, 1)
	// beta2 = max(gamma, nu/sqrt(n^^2-1), eps)

	for (int j = 0; j < 4; ++j) { // compute column j
		
		for (int s = 0; s < j; ++s)
			mat[j*4+s] /= mat[s*4+s];
		for (int i = j + 1; i < 4; ++i) {
			t0 = mat[i*4+j];
			for (int s = 0; s < j; ++s)
				t0 -= mat[j*4+s] * mat[i*4+s];
			mat[i*4+j] = t0;
		}

		t1 = 0.0f;
		for (int i = j + 1; i < 4; ++i) {
			t0 = fabsf(mat[i*4+j]);
			if (t1 < t0)
				t1 = t0;
		}
		t1 *= t1;

		t2 = fabsf(mat[j*4+j]);
		if (t2 < delta)
			t2 = delta;
		t0 = t1 / beta2;
		if (t2 < t0)
			t2 = t0;
		mat[j*4+j] = t2;

		if (j < 4) {
			for (int i = j + 1; i < 4; ++i) {
				t0 = mat[i*4+j];
				mat[i*4+i] -= t0*t0/t2;
			}
		}

	}

}

__device__
void permute_vec_4_f(const float* vec, const int* perm, float* ovec) 
{
	for (int i = 0; i < 4; ++i) {
		ovec[i] = vec[perm[i]];
	}
}

__device__
void forward_subs_unit_diaged_4_f(const float* mat, const float* rhs, float* sol) 
{
	for (int i = 0; i < 4; ++i) {
		sol[i] = rhs[i];
		for (int j = 0; j < i; ++j) {
			sol[i] -= mat[i*4+j] * mat[j*4+j] * sol[j];
		}
		sol[i] /= mat[i*4+i];
	}
}

__device__
void backward_subs_unit_t_4_f(const float* mat, const float* rhs, float* sol) 
{
	for (int i = 4 - 1; i >= 0; --i) {
		sol[i] = rhs[i];
		for (int j = i + 1; j < 4; ++j) {
			sol[i] -= mat[j*4+i] * sol[j];
		}
	}
}

__device__
void ldl_solve_4_f(const float* mat, const float* rhs, float* sol) 
{
	float arr[4];
	forward_subs_unit_diaged_4_f(mat, rhs, arr);
	backward_subs_unit_t_4_f(mat, arr, sol);
}

__device__
void inv_permute_vec_4_f(const float* vec, const int* perm, float* ovec) 
{
	for (int i = 0; i < 4; ++i) {
		ovec[perm[i]] = vec[i];
	}
}

__device__
void gmw81_solver_4_f(float* mat, const float* rhs, float* sol) 
{	
	// Diagonal pivoting of matrix and right hand side
	int perm[4];
	float arr1[4];
	float arr2[4];
	diag_pivot_4_f(mat, perm);
	permute_vec_4_f(rhs, perm, arr1);
	
	// Diagonaly scale the matrix and rhs to improve condition number
	float scale[4];
	for (int i = 0; i < 4; ++i) {
		scale[i] = sqrtf(fabsf(mat[i*4+i]));
		arr1[i] /= scale[i];
	}
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j <= i; ++j) {
			mat[i*4+j] /= (scale[i] * scale[j]);
		}
	}

	gmw81_4_f(mat);
	ldl_solve_4_f(mat, arr1, arr2);

	// Unscale
	for (int i = 0; i < 4; ++i) {
		arr2[i] /= scale[i];
	}

	// Unpivot solution
	inv_permute_vec_4_f(arr2, perm, sol);
}

__device__
void gain_ratio_step_4_f(const float* f, const float* ftp, const float* pars_tp, const float* step,
	const float* g, const float* h, float* pars, 
	float* lam, char* step_type, float mu, float eta, float acc, float dec, int tid, int Nprobs) 
{

	float actual = 0.5f * (f[tid] - ftp[tid]);
	float predicted = 0.0f;

	int k = 0;
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j <= i; ++j) {
			float entry = h[k*Nprobs+tid] * step[i*Nprobs+tid] * step[j*Nprobs+tid];
			if (i == j) {
				predicted -= entry;
			} else {
				predicted -= 2.0f * entry;
			}
			++k;
		}
	}
	predicted *= 0.5f;

	for (int i = 0; i < 4; ++i) {
		int iidx = i*Nprobs+tid;
		predicted += step[iidx] * g[iidx];
	}

	float rho = actual / predicted;

	if ((rho > mu) && (actual > 0)) {
		for (int i = 0; i < 4; ++i) {
			int iidx = i*Nprobs+tid;
			pars[iidx] = pars_tp[iidx];
			if (tid == 0) {
				printf("pars copied ");
			}
		}
		if (rho > eta) {
			lam[tid] *= acc;
			step_type[tid] = 1;
		} else {
			step_type[tid] = 2;
		}
	} else {
		lam[tid] *= dec;
		step_type[tid] = 4;
	}

	if (predicted < 0) {
		lam[tid] *= dec;
		step_type[tid] |= 8;
	}

	if (tid == 0) {
		printf(" rho=%f, actual=%f, f=%f, \n", rho, actual, f[tid]);
	}

}

__device__
void clamp_pars_4_f(const float* lower_bound, const float* upper_bound, float* pars, int tid, int N) 
{
	for (int i = 0; i < 4; ++i) {
		int index = i*N+tid;
		float p = pars[index];
		float u = upper_bound[index];
		float l = lower_bound[index];

		if (p > u) {
			pars[index] = u;
		} else if (p < l) {
			pars[index] = l;
		}
	}
}

__device__
void gradient_convergence_4_f(const float* pars, const float* g, const float* f, const float* lower_bound, const float* upper_bound, char* step_type, float tol, int tid, int N) 
{
	bool clamped = false;
	float clamped_norm = 0.0f;
	float temp1;
	float temp2;
	for (int i = 0; i < 4; ++i) {
		int iidx = i*N+tid;
		temp1 = pars[iidx];
		temp2 = g[iidx];
		temp2 = temp1 - temp2;
		float u = upper_bound[iidx];
		float l = lower_bound[iidx];
		if (temp2 > u) {
			clamped = true;
			temp2 = u;
		} else if (temp2 < l) {
			clamped = true;
			temp2 = l;
		}
		temp2 = temp1 - temp2;
		clamped_norm += temp2*temp2;
	}

	if (clamped_norm < tol*(1 + f[tid])) {
		if ((step_type[tid] & 1) || clamped) {
			step_type[tid] = 0;
		}
	}
}

__device__
void second_order_levenberg_marquardt_4_0_9_f_enc_vel_fgradf(const float* consts, const float* data, const float* lower_bound, const float* upper_bound, 
	float tol, float mu, float eta, float acc, float dec,
	float* params, float* params_tp, float* step, float* lam, char* step_type,
	float* f, float* ftp, float* g, float* h, float* hl, int tid, int Nprobs)
{
	// Set gradients and objective functions to zero
	{
		f[tid] = 0.0f;
		ftp[tid] = 0.0f;
		for (int i = 0; i < 4; ++i) {
			g[i*Nprobs+tid] = 0.0f;
		}
		for (int i = 0; i < 10; ++i) {
			h[i*Nprobs+tid] = 0.0f;
			hl[i*Nprobs+tid] = 0.0f;
		}
	}

	// Calculate gradients
	{
		enc_to_vel_fghhl(params, consts, data, lam, f, g, h, hl, tid, Nprobs);
	}

	// Solve step
	{
		float* mat = hl;
		float* rhs = g;
		float* sol = step;

		float mat_copy[4*4];
		float rhs_copy[4];
		float sol_copy[4];

		for (int i = 0; i < 4; ++i) {
			rhs_copy[i] = rhs[i*Nprobs+tid];
			sol_copy[i] = sol[i*Nprobs+tid];
		}
		int k = 0;
		for (int i = 0; i < 4; ++i) {
			for (int j = 0; j <= i; ++j) {
				float temp = mat[k*Nprobs+tid];
				mat_copy[i*4+j] = temp;
				if (i != j) {
					mat_copy[j*4+i] = temp;
				}
				++k;
			}
		}

		gmw81_solver_4_f(mat_copy, rhs_copy, sol_copy);

		for (int i = 0; i < 4; ++i) {
			sol[i*Nprobs+tid] = sol_copy[i];
		}
	}

	// Remove NaN and Infs
	{
		for (int i = 0; i < 4; ++i) {
			int idx = i*Nprobs+tid;

			// Remove inf
			float si = step[idx];
			if (isnan(si) || isinf(si)) {
				step[idx] = 0.0f;
			}
		}
	}

	// Check convergence
	{
		gradient_convergence_4_f(params, g, f, lower_bound, upper_bound, step_type, tol, tid, Nprobs);
		if (step_type[tid] == 0) {
			return;
		}
	}

	// Subtract step from params
	{
		for (int i = 0; i < 4; ++i) {
			int idx = i*Nprobs+tid;
			params_tp[idx] = params[idx] - step[idx];
		}
	}

	// Calculate error at new params
	{
		enc_to_vel_f(params_tp, consts, data, ftp, tid, Nprobs);
	}

	// Calculate gain ratio and determine step type
	{
		gain_ratio_step_4_f(f, ftp, params_tp, step, g, h, params, lam, step_type, mu, eta, acc, dec, tid, Nprobs);
	}

	// Clamp parameters to feasible region
	{
		clamp_pars_4_f(lower_bound, upper_bound, params, tid, Nprobs);
	}

}

extern "C" __global__
void k_second_order_levenberg_marquardt_4_0_9_f_enc_vel_fgradf(const float* consts, const float* data, const float* lower_bound, const float* upper_bound,
	float tol, float mu, float eta, float acc, float dec,
	float* params, float* params_tp, float* step, float* lam, char* step_type,
	float* f, float* ftp, float* g, float* h, float* hl, int Nprobs) 
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < Nprobs) {

		if (step_type[tid] == 0) {
			return;
		}

		second_order_levenberg_marquardt_4_0_9_f_enc_vel_fgradf(consts, data, lower_bound, upper_bound, tol, mu, eta, acc, dec, params, params_tp, step, lam, step_type, f, ftp, g, h, hl, tid, Nprobs);		
	}
}
